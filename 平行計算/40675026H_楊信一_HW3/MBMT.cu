#include<stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


#define NUM_ROWS 10000
#define NUM_COLS 1000
int	ha[NUM_ROWS][NUM_COLS] ;
int	hb[NUM_ROWS][NUM_COLS] ;
int	hc[NUM_ROWS][NUM_COLS] ;

 __global__ void add(int* da, int* db, int* dc){
    int tid = blockDim.x * blockDim.y * (blockIdx.y * gridDim.x + blockIdx.x) + (threadIdx.y * blockDim.x + threadIdx.x);
	while(tid<NUM_ROWS* NUM_COLS){
        dc[tid] = da[tid]+ db[tid];
        tid= tid + blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    }
}

int main(){
    int	*da;
    int *db;
    int *dc;
    int iSize = NUM_ROWS * NUM_COLS * sizeof(int) ;
    hipError_t     cuError = hipSuccess;
	int total=NUM_ROWS*NUM_COLS/512;
    dim3 dimGrid (total+1, 1, 1) ;
    dim3 dimBlock (512, 1, 1) ;

    for(int i=0;i<NUM_ROWS;i++){
        for(int j=0;j<NUM_COLS;j++){
            ha[i][j]=rand()%10+1;
	        hb[i][j]=rand()%10+1;
        }
    }

    cuError = hipMalloc((void**)&da, iSize) ;
    if (hipSuccess != cuError){
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }
    cuError = hipMemcpy(da, ha, iSize, hipMemcpyHostToDevice);
    if (hipSuccess != cuError){
        hipFree (da) ;
        printf ("Failed in Memcpy 1\n") ;
        return 1 ;
    }
	
    cuError = hipMalloc((void**)&db, iSize) ;
    if (hipSuccess != cuError){
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }
    cuError = hipMemcpy(db, hb, iSize, hipMemcpyHostToDevice);
    if (hipSuccess != cuError){
        hipFree (db) ;
        printf ("Failed in Memcpy 1\n") ;
        return 1 ;
    }
    cuError = hipMalloc((void**)&dc, iSize) ;
    if (hipSuccess != cuError){
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }


	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    add<<<dimGrid, dimBlock>>>(da, db, dc);
	
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
	
	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
    cuError = hipGetLastError () ;
    if (hipSuccess != cuError){
        printf ("Failed in kernel launch and reason is %s\n", hipGetErrorString(cuError)) ;
        return 1 ;
    }

    cuError = hipMemcpy(hc, dc, iSize, hipMemcpyDeviceToHost);
    if (hipSuccess != cuError){
        hipFree (dc) ;
        printf ("Failed in Memcpy 2\n") ;
        return 1 ;
    }

    bool success = true;
    for(int i=0;i<NUM_ROWS;i++){
        for(int j=0;j<NUM_COLS;j++){
            if ((ha[i][j] + hb[i][j]) != hc[i][j]){
		        printf( "Error:  %d + %d != %d\n", ha[i][j], hb[i][j], hc[i][j] );
		        success = false;
	        } 
	    }
    }
    if (success) printf( "We did it!\n" );
	
    hipFree (da) ;
    hipFree (db) ;	
    hipFree (dc) ;

    return 0;
}