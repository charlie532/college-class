#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define N 1000000000
#define KERNELSIZE 9
#define THREADSPERBLOCK 1024
#define BLOCKSPERGRID (N+THREADSPERBLOCK-1)/THREADSPERBLOCK


// 9 wide 1d kernel, no padding so it cuts out early
// shift by 4 to align with original data
__global__ void conv( float *data, float *kernel, float *output ){
    int tid =blockIdx.x*blockDim.x+threadIdx.x;
	int i;

	for(i=0; i<9; i++){
		output[tid] += data[tid + i] * kernel[i];
	}
}

int main(){
	srand(time(NULL));
    struct timespec t_start, t_end;
	double elapsedTimeCPU;
	int pass = 1;

	// gassian kernel from: http://dev.theomader.com/gaussian-kernel-calculator/
	float kernel[9] = {0.000229, 0.005977, 0.060598, 0.241732, 0.382928, 0.241732, 0.060598, 0.005977, 0.000229};


	// random number from python, irl this would come from lidar
	float* data = (float*)malloc(N*sizeof(float));
	for(int i = 0;i < N;i++){
		data[i] = rand()%10+rand()/RAND_MAX;
	}

	// empty array to store the output
	// float output[N-KERNELSIZE+1];

	//CPU
    clock_gettime( CLOCK_REALTIME, &t_start); 
	float* output = (float*)malloc((N-KERNELSIZE+1)*sizeof(float));
	for (int i = 0; i < N-KERNELSIZE+1;i++){
		output[i] = 0;
		for (int j = 0; j < KERNELSIZE; j++){
			output[i] += kernel[j] * data[i+j];
		}
	}
    clock_gettime( CLOCK_REALTIME, &t_end);
    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
	elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);


	//GPU
	float *d_kernel, *d_data, *d_output;
	// allocate the memory on the GPU
    hipMalloc( (void**)&d_kernel, KERNELSIZE * sizeof(float) );
    hipMalloc( (void**)&d_data, N * sizeof(float) );
	hipMalloc( (void**)&d_output, (N-KERNELSIZE+1) * sizeof(float) );
	float* output_from_device = (float*)malloc((N-KERNELSIZE+1)*sizeof(float));
	hipMemcpy( d_kernel, kernel, KERNELSIZE * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_data, data, N * sizeof(float), hipMemcpyHostToDevice );

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	conv<<<BLOCKSPERGRID, THREADSPERBLOCK>>>(d_data, d_kernel, d_output);
  	hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
	hipMemcpy(output_from_device, d_output, (N-KERNELSIZE+1) * sizeof(float), hipMemcpyDeviceToHost );

	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU elapsedTime: %lf ms\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("speedup: %lf \n", elapsedTimeCPU/elapsedTime);
	for (int i = 0; i < N-KERNELSIZE+1; i++){
		if(output_from_device[i]-output[i]>0.00001){ //don't use if(output_from_device[i]!=output[i])
			printf("CPU:%lf    GPU:%lf\n",output[i], output_from_device[i] );
			pass = 0;
		}
	}
	if(pass == 1)
		printf("Test pass!\n");
	else
		printf("Test fail!\n");
}