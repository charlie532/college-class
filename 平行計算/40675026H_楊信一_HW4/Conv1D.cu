#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define N 1024*1024*1024
#define KERNELSIZE 9
#define THREADSPERBLOCK 1024
#define BLOCKSPERGRID (N+THREADSPERBLOCK-1)/THREADSPERBLOCK


// 9 wide 1d kernel, no padding so it cuts out early
// shift by 4 to align with original data
__global__ void conv( float *data, float *kernel, float *output ){
    int tid =blockIdx.x*blockDim.x+threadIdx.x;
	int i;

	for(i=0; i<9; i++){
		output[tid] += data[tid + i] * kernel[i];
	}

}

int main(){
	srand(time(NULL));
	
    struct timespec t_start, t_end;
	double elapsedTimeCPU;
	int pass = 1;

	// gassian kernel from: http://dev.theomader.com/gaussian-kernel-calculator/
	float kernel[9] = {0.000229, 0.005977, 0.060598, 0.241732, 0.382928, 0.241732, 0.060598, 0.005977, 0.000229};


	// random number from python, irl this would come from lidar
	//float data[100] = {7.230, 16.98, 17.99, 1.703, 16.44, 4.484, 7.843, 13.44, 7.815, 11.91, 2.050, 6.138, 3.049, 0.167, 1.756, 10.46, 10.02, 10.48, 13.14, 7.329, 14.93, 7.275, 18.61, 13.82, 15.97, 11.43, 10.27, 5.290, 14.13, 2.671, 3.267, 6.149, 14.56, 13.11, 18.14, 16.47, 17.49, 16.20, 7.835, 5.883, 0.967, 0.237, 4.359, 13.15, 15.92, 16.94, 14.30, 17.47, 5.118, 5.142, 19.41, 5.046, 16.78, 3.944, 12.17, 7.983, 15.35, 7.839, 11.65, 12.56, 9.564, 14.30, 4.670, 1.893, 9.304, 0.173, 3.921, 15.63, 6.561, 16.25, 1.634, 4.870, 15.03, 0.269, 11.92, 0.390, 15.57, 2.918, 8.966, 14.04, 11.23, 7.519, 7.943, 6.570, 18.74, 15.54, 1.303, 14.01, 1.797, 1.526, 12.90, 3.051, 8.602, 7.094, 14.39, 14.13, 11.20, 2.637, 2.644, 2.810};
	float* data = (float*)malloc(N*sizeof(float));
	for(int i = 0;i < N;i++)
	{
		data[i] = rand()%10+rand()/RAND_MAX;
	}

	// empty array to store the output
	// float output[N-KERNELSIZE+1];

	//CPU 1d convolutional operation

    clock_gettime( CLOCK_REALTIME, &t_start); 

	float* output = (float*)malloc((N-KERNELSIZE+1)*sizeof(float));
	for (int i = 0; i < N-KERNELSIZE+1;i++)
	{
		output[i] = 0;
		for (int j = 0; j < KERNELSIZE; j++){
			output[i] += kernel[j] * data[i+j];
		}
	}

    clock_gettime( CLOCK_REALTIME, &t_end);

    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
	elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);


	//GPU
	float *d_kernel, *d_data, *d_output;
	// allocate the memory on the GPU
    hipMalloc( (void**)&d_kernel, KERNELSIZE * sizeof(float) );
    hipMalloc( (void**)&d_data, N * sizeof(float) );
	hipMalloc( (void**)&d_output, (N-KERNELSIZE+1) * sizeof(float) );

	float* output_from_device = (float*)malloc((N-KERNELSIZE+1)*sizeof(float));
	hipMemcpy( d_kernel, kernel, KERNELSIZE * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_data, data, N * sizeof(float), hipMemcpyHostToDevice );

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);



	conv<<<BLOCKSPERGRID, THREADSPERBLOCK>>>(d_data, d_kernel, d_output);
  	hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
	hipMemcpy(output_from_device, d_output, (N-KERNELSIZE+1) * sizeof(float), hipMemcpyDeviceToHost );

	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("加速 %13f 倍\n", elapsedTimeCPU/elapsedTime);
	for (int i = 0; i < N-KERNELSIZE+1; i++){
		if(output_from_device[i]-output[i]>0.00001){ //don't use if(output_from_device[i]!=output[i])
			printf("CPU:%lf    GPU:%lf\n",output[i], output_from_device[i] );
			pass = 0;
		}
	}

	if(pass == 1)
		printf("Test pass!\n");
	else
		printf("Test fail!\n");
}