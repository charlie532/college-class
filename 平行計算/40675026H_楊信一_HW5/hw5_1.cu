#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 1024

__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width);

    float M[WIDTH][WIDTH] = {0};
    float N[WIDTH][WIDTH] = {0};
    float P[WIDTH][WIDTH] = {0};
    float MxN[WIDTH][WIDTH] = {0};
int main(int argc, char *argv[])
{
    int width = WIDTH;
    int pass = 1;
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            M[i][j] = rand() % 30;
            N[i][j] = rand() % 30;
        }
    }
    
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            for (int k = 0; k < width; ++k) {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);
    
	size_t size = width * width * sizeof(float);
    float *Md, *Nd, *Pd;
    
    // Allocate and Load M, N to device memory
    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    
    // Allocate P on the device
    hipMalloc((void **)&Pd, size);
    
    // Setup the execution configuration
    dim3 dimGrid(1, 1);
    dim3 dimBlock(32, 32);
    
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            for (int k = 0; k < width; ++k) {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    // Invoke kernel
    MatMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);
	
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
	printf("speedup:%lf\n",executime/elapsedTime);
    // Read P from device memory
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            if(MxN[i][j] != P[i][j]) {
		printf("MxN[%d][%d] = %2.0f   P[%d][%d] = %2.0f\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
            }
        }
    }
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");
    
    return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
    // Thread row and column within matrix
    int i = threadIdx.y;//i is row
    int j = threadIdx.x;//j is column
    
    // Each thread computes one element of P
    // by accumulating results into Pvalue
    float Pvalue;
    
    // Multiply M and N
    while(i < width){
		j = threadIdx.x;
		while(j < width ){
			Pvalue = 0;
			for (int k = 0; k < width; ++k) {
				float Melement = *(Md + i*width + k);
				float Nelement = *(Nd + k*width + j);
				Pvalue += Melement * Nelement;
			}
			*(Pd + i*width + j) = Pvalue;
			j = j + blockDim.x;
		}
		i = i + blockDim.y;
    }
	
}