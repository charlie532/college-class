#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define WIDTH 1024  // 64 ~ 512
#define TILE_WIDTH  32



__device__ float GetElement(float *matrix, int row, int col, int width);
__device__ void SetElement(float *matrix, int row, int col, int width, float value);
__device__ float *GetSubMatrix(float *matrix, int blockrow, int blockcol, int width);
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width);

float M[WIDTH][WIDTH] = {0};
float N[WIDTH][WIDTH] = {0};  
float P[WIDTH][WIDTH] = {0};
float MxN[WIDTH][WIDTH] = {0};
int main(int argc, char *argv[])
{
    int width = WIDTH;
    int pass = 1;
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            M[i][j] = rand() % 30;
            N[i][j] = rand() % 30;
        }
    }
    
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            for (int k = 0; k < width; ++k) {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);
    
    size_t size = width * width * sizeof(float);
    float *Md, *Nd, *Pd;
    
    // Allocate and Load M, N to device memory
    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    
    // Allocate P on the device
    hipMalloc((void **)&Pd, size);
    
    // Setup the execution configuration
    dim3 dimGrid(width/TILE_WIDTH, width/TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // Invoke kernel
    MatMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("speedup:%lf\n",executime/elapsedTime);
    // Read P from device memory
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);   
   
   
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            if(MxN[i][j] != P[i][j]) {
		printf("MxN[%d][%d] = %2.0f   P[%d][%d] = %2.0f\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
            }
        }
    }
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");
    
    return 0;
}

// Get a matrix element
__device__ float GetElement(float *matrix, int row, int col, int width)
{
    return *(matrix + row*width + col);
}

// Set a matrix element
__device__ void SetElement(float *matrix, int row, int col, int width, float value)
{
    *(matrix + row*width + col) = value;
}

// Get the TILE_WIDTHxTILE_WIDTH sub-matrix matsub of matrix that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of matrix
__device__ float *GetSubMatrix(float *matrix, int blockrow, int blockcol, int width)
{
    return (matrix + blockrow*TILE_WIDTH*width + blockcol*TILE_WIDTH);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    
    // Each thread block computes one sub-matrix Psub of P
    float *Pd_sub = GetSubMatrix(Pd, blockRow, blockCol, width);
    
    // Thread row and column within sub-matrix
    int row = threadIdx.y;
    int col = threadIdx.x;
    
    // Each thread computes one element of Psub
    // by accumulating results into Pvalue
    float Pvalue = 0;
    
    // Loop over all the sub-matrices of M and N that are
    // required to compute Psub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (width / TILE_WIDTH); ++m) {
        // Get sub-matrix Msub of M
        float *Md_sub = GetSubMatrix(Md, blockRow, m, width);
        // Get sub-matrix Nsub of N
        float *Nd_sub = GetSubMatrix(Nd, m, blockCol, width);
        
        // Multiply Msub and Nsub together
        for (int k = 0; k < TILE_WIDTH; ++k) {
            float Melement = GetElement(Md_sub, row, k, width);
            float Nelement = GetElement(Nd_sub, k, col, width);
            Pvalue += Melement * Nelement;
        }
        
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of M and N in the next iteration
        __syncthreads();
    }
    
    // Write Psub to device memory
    // Each thread writes one element
    SetElement(Pd_sub, row, col, width, Pvalue);
}
